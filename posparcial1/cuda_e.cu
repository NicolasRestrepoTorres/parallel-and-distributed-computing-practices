
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square( int *d_num_steps, unsigned long long *d_fact, double *d_out){
  int idx = threadIdx.x;
  int num_steps = *d_num_steps;
  for(int k=idx+1; k< num_steps; k+=blockDim.x){
    d_out[idx] += (double) k*0.5/ (double) d_fact[k-1];
  }

}

int main(int argc, char ** argv){
  int h_num_steps = 21;
  int THREADS = atoi(argv[1]);
  double e;

  // generate the output array on the host
  unsigned long long h_fact[h_num_steps];
  h_fact[0]=1;
  for(int f=1; f<h_num_steps; f++){
    h_fact[f] = h_fact[f-1]*f;
  }
  double h_out[THREADS];

  //declare GPU memory pointers
  int *d_num_steps;
  unsigned long long *d_fact;
  double *d_out;

  //allocate GPU memory
  hipMalloc((void **) &d_num_steps, sizeof(int));
  hipMalloc((void **) &d_fact, h_num_steps*sizeof(unsigned long long));
  hipMalloc((void **) &d_out, THREADS*sizeof(double));

  // transfer the array to the GPU
  hipMemcpy(d_num_steps, &h_num_steps, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_fact, h_fact, h_num_steps*sizeof(unsigned long long), hipMemcpyHostToDevice);

  // launch the kernel
  square<<<1, THREADS>>>(d_num_steps, d_fact, d_out);

  //copy back the result array to the CPU
  hipMemcpy(h_out, d_out, THREADS*sizeof(double), hipMemcpyDeviceToHost);

  //print out the resulting array
  for (int i=0; i<THREADS; i++){
    e+=h_out[i];
  }

  printf("e: %f\n", e);

  // free GPU memory allocation
  hipFree(d_num_steps);
  hipFree(d_fact);
  hipFree(d_out);

  return 0;
}
