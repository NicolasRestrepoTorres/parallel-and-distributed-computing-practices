#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <bits/stdc++.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Lock.h"

using namespace cv;
using namespace std;
struct timeval  tv1, tv2;
Mat img, blurred;
//Compile with nvcc cudaBlur.cu `pkg-config opencv --cflags --libs`

__global__ void boxBlur(uchar4 *d_img, uchar4 *d_blurred, int numRows, int numCols, int k){


	printf("Hilo: %d\n", threadIdx.x);
	printf("KERNEL PAANIC R: %d, G: %d, B: %d\n", d_img[numCols*69+45].x, d_img[numCols*1 +7].y, d_img[numCols*1 +7].z);

	int id = threadIdx.x;
	int accumulator, cornerX, cornerY, sumRGB[] = {0,0,0};


		for(int x=id; x<numRows; x+=blockDim.x){
			//printf("hilo: %d, col: %d, blockDim.x: %d\n", id, x, blockDim.x);

			for(int y=0; y<numCols; y++){
				//printf("hilo: %d, row: %d, col: %d, blockDim.x: %d\n", id, x, y, blockDim.x);
				//printf("(%d,%d,%d) R: %d, G: %d, B: %d\n", id, x, y, d_img[numCols*x+y].x, d_img[numCols*x+y].y, d_img[numCols*x+y].z);
				//cout << y << endl;

        int pixel = numCols * x + y;
        uchar4 pixelColor = d_img[pixel];
				//printf("R: %d, G: %d, B: %d\n", pixelColor.x, pixelColor.y, pixelColor.z);
				sumRGB[0]=0;
				sumRGB[1]=0;
				sumRGB[2]=0;
				accumulator=0;
				if(k%2==0){
					cornerX = x-(k-1)/2;
					cornerY = y-(k-1)/2;
				}else{
					cornerX = x-(k-2)/2;
					cornerY = y-(k-2)/2;
				}

				for(int i = 0; i < k; i++){
					for(int j = 0; j < k; j++){
            if(cornerX+i>=0 && cornerX+i<numRows && cornerY+j>=0 && cornerY+j<numCols) {
              int index = numCols * (cornerX+i) + (cornerY+j);
							uchar4 color = d_img[index];
							sumRGB[0] += (int) color.x;
							sumRGB[1] += (int) color.y;
							sumRGB[2] += (int) color.z;
							accumulator++;
						}
					}
				}



				if(accumulator == 0){
					printf("WHAAAA?\n" );
					sumRGB[0] = pixelColor.x;
					sumRGB[1] = pixelColor.y;
					sumRGB[2] = pixelColor.z;
				}else{
					sumRGB[0] = sumRGB[0] / accumulator;
					sumRGB[1] = sumRGB[1] / accumulator;
					sumRGB[2] = sumRGB[2] / accumulator;
				}

				uchar4 color;
				color.x = sumRGB[0];
				color.y = sumRGB[1];
				color.z = sumRGB[2];

				//myLock.lock();
				d_blurred[pixel] = color;
				//myLock.unlock();
				//printf("Original R: %d, G: %d, B: %d\n", pixelColor.x, pixelColor.y, pixelColor.z);
				//printf("Blurred  R: %d, G: %d, B: %d\n", d_blurred[pixel].x, d_blurred[pixel].y, d_blurred[pixel].z);

			}

		}


}

int main(int argc, char** argv ){
		//Lock myLock;

    img = imread( argv[1], 1 );
		if ( !img.data ){
        printf("No image data \n");
        return -1;
    }

		blurred = Mat(img.rows, img.cols, CV_8UC4);

    size_t numRows =  img.rows;
    size_t numCols = img.cols;
    const size_t numPixels = numRows * numCols;

		size_t k = atoi(argv[3]);
		int THREADS = atoi(argv[4]);

		printf("rows %d, cols %d\n", img.rows, img.cols);


    uchar4 *h_img = (uchar4 *)img.ptr<unsigned char>(0);
    uchar4 *h_blurred = (uchar4 *)blurred.ptr<unsigned char>(0);
		uchar4 *d_img;
    uchar4 *d_blurred;
		printf("NN R: %d, G: %d, B: %d\n", h_img[img.cols*69+45].x, h_img[img.cols*1 +7].y, h_img[img.cols*1 +7].z);

    //allocate memory on the device for both input and output
		hipMalloc((void **) &d_img, numPixels * sizeof(uchar4));
  	hipMalloc((void **) &d_blurred, numPixels * sizeof(uchar4));
		printf("1\n");
    //make sure no memory is left laying around
    hipMemset(d_blurred, 0, numPixels * sizeof(uchar4));
		printf("2\n");
    //copy input array to the GPU
    hipMemcpy((void *)d_img, (void *)h_img, numPixels * sizeof(uchar4), hipMemcpyHostToDevice);
		printf("3\n");
		boxBlur<<<1, THREADS>>>(d_img, d_blurred, numRows, numCols, k);
		printf("4\n");
		hipMemcpy((void *)h_blurred, (void *)d_blurred, numPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
		printf("5\n");
		printf("BLURRED R: %d, G: %d, B: %d\n", h_blurred[numCols*69+45].x, h_blurred[numCols*1 +7].y, h_blurred[numCols*1 +7].z);

		Mat output = Mat(numRows, numCols, CV_8UC4, (void*)h_blurred);
		Mat blu;
	  cvtColor(output, blu, CV_RGBA2BGR);
		imwrite( argv[2], blu);

		hipFree(d_img);
  	hipFree(d_blurred);


		return 0;
}
