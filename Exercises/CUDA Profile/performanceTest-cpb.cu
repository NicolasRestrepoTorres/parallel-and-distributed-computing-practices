/*programa para hacer un calculo intensivo y hacer nvprof*/

#include <stdio.h>

#include <hip/hip_runtime.h>
#define ITERATIONS 1E07

__global__ void calcSin(float *A)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i;
    double acc, ang = 10.0;
      for(i = 0; i < ITERATIONS; i++){
        ang = i;
        acc = sin(ang);
      }
      *A = acc;
}

__global__ void calcAdd(float *A)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i;
    double acc, ang = 10.0;
      for(i = 0; i < ITERATIONS; i++){
        acc = acc + ang;
      }
      *A = acc;
}

__global__ void calcMul(float *A)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i;
    double acc = 1.0, ang = 13.0;
      for(i = 0; i < ITERATIONS; i++){
        acc = acc * i;
      }
      *A = acc;
}

int main(void)
{
    hipError_t err = hipSuccess;

    printf("Calculating...");
    float *d_A = NULL;
    float A;
    err = hipMalloc((void **)&d_A, sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 48;
    int blocksPerGrid = 2;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    calcSin<<<blocksPerGrid, threadsPerBlock>>>(d_A);
    calcAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A);
    calcMul<<<blocksPerGrid, threadsPerBlock>>>(d_A);
	hipDeviceSynchronize();
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(&A, d_A, sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("\n%f ", A);
    err = hipFree(d_A);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}
