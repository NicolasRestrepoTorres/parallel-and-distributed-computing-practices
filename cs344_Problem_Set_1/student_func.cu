#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"
#include <stdio.h>



__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       uchar4 * const greyImage,
                       int numRows, int numCols)
{
  //TODO (solved)
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D index
  // could also use blockDim.x instead of numCols
  //Columna, fila
  int index = numCols * blockIdx.x + threadIdx.x;
  uchar4 rgbpx = rgbaImage[index];
  // greyImage[index] = (unsigned char)(0.299f * rgbpx.x + 0.587f *
  //   rgbpx.y + 0.114f * rgbpx.z);
    int size_kernel = 3;
    int start_x = blockIdx.x; int start_y = threadIdx.x;
    int accumulator = 0; double red = 0.0; double green = 0.0; double blue = 0.0;
    int current_pixel; bool valid;
    uchar4 rgbpx_current_pixel;
    for(int i = start_x; i < (start_x + size_kernel); i++){
      for(int j = start_y; j < (start_y + size_kernel); j++){
        //
        if (i < 0 || i >= numCols) valid = false;
        else if (j < 0 || j >= numRows) valid = false;
        else valid = true;
        if (valid){
          current_pixel = numCols * i + j;
          rgbpx_current_pixel = rgbaImage[current_pixel];
          red += rgbpx_current_pixel.x;
          green += rgbpx_current_pixel.y;
          blue += rgbpx_current_pixel.z;

          // greyImage[index] = (unsigned char)(0.299f * rgbpx_current_pixel.x +
          //   0.587f * rgbpx_current_pixel.y + 0.114f * rgbpx_current_pixel.z);
            accumulator += 1;
        }
      }
    }


    if(index % 10000 == 0){
    printf("Original %d: %d \n", index, rgbpx.x);
    rgbpx.x = red;
    rgbpx.y = green;
    rgbpx.z = blue;
    printf("Changed %d: %d \n",index, rgbpx.x);

    greyImage[index] =   rgbpx;
    }

    //
    //
  	// int debug = 0;
  	// int id = index;
    //
    //
  	// int accumulator, cornerX, cornerY, sumRGB[] = {0,0,0};
    //
  	// 	//printf("Hilo: %d\n", id);
  	// 	for(int x=id;x<img.cols;x+=THREADS){
  	// 		//printf("hilo: %d, col: %d\n", id, x);
  	// 		for(int y=0;y<img.rows;y++){
  	// 			//sem_wait(&semvar2);
  	// 			//cout << y << endl;
    //
  	// 			sumRGB[0]=0;
  	// 			sumRGB[1]=0;
  	// 			sumRGB[2]=0;
  	// 			accumulator=0;
  	// 			if(k%2==0){
  	// 				cornerX = x-(k-1)/2;
  	// 				cornerY = y-(k-1)/2;
  	// 			}else{
  	// 				cornerX = x-(k-2)/2;
  	// 				cornerY = y-(k-2)/2;
  	// 			}
    //
  	// 			for(int i = 0; i < k; i++){
  	// 				for(int j = 0; j < k; j++){
  	// 					if(checkBounds(cornerX+j,cornerY+i, img.cols, img.rows)){
  	// 						Vec3b color = img.at<Vec3b>(Point(cornerX+j,cornerY+i));
  	// 						sumRGB[0] += (int) color(0);
  	// 						sumRGB[1] += (int) color(1);
  	// 						sumRGB[2] += (int) color(2);
  	// 						accumulator++;
  	// 					}
  	// 				}
  	// 			}
    //
  	// 			if(accumulator == 0){
  	// 				sumRGB[0] = img.at<Vec3b>(Point(x,y))(0);
  	// 				sumRGB[1] = img.at<Vec3b>(Point(x,y))(1);
  	// 				sumRGB[2] = img.at<Vec3b>(Point(x,y))(2);
  	// 			}else{
  	// 				sumRGB[0] = sumRGB[0] / accumulator;
  	// 				sumRGB[1] = sumRGB[1] / accumulator;
  	// 				sumRGB[2] = sumRGB[2] / accumulator;
  	// 			}
  	// 			//sem_post(&semvar2);
  	// 			Vec3b color;
  	// 			color(0) = sumRGB[0];
  	// 			color(1) = sumRGB[1];
  	// 			color(2) = sumRGB[2];
  	// 			#pragma omp critical
  	// 				blurred.at<Vec3b>(Point(x,y)) = color;
    //
    //
  	// 		}
  	// 	}

}




void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            uchar4 * const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 gridSize(numRows, 1, 1);  //TODO (solved)
  const dim3 blockSize(numCols, 1, 1);  //TODO (solved)
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
