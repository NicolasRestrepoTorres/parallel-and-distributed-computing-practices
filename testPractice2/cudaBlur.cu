#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <bits/stdc++.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace cv;
using namespace std;
struct timeval  tv1, tv2;
Mat img, blurred;
//Compile with nvcc cudaBlur.cu `pkg-config opencv --cflags --libs`

__global__ void BoxBlur(uchar4 *d_img, uchar4 *d_blurred, int numRows, int numCols, int k){

	int id = threadIdx.x;
	int accumulator, cornerX, cornerY, sumRGB[] = {0,0,0};

		printf("Hilo: %d\n", id);
		for(int x=id; x<numRows; x+=blockDim.x){
			//printf("hilo: %d, col: %d\n", id, x);
			for(int y=0; y<numCols; y++){
				//sem_wait(&semvar2);
				//cout << y << endl;
        int pixel = numCols * x + y;
        uchar4 pixelColor = d_img[pixel];
				sumRGB[0]=0;
				sumRGB[1]=0;
				sumRGB[2]=0;
				accumulator=0;
				if(k%2==0){
					cornerX = x-(k-1)/2;
					cornerY = y-(k-1)/2;
				}else{
					cornerX = x-(k-2)/2;
					cornerY = y-(k-2)/2;
				}

				for(int i = 0; i < k; i++){
					for(int j = 0; j < k; j++){
            if(cornerX+i>=0 && cornerX+i<numRows && cornerY+j>=0 && cornerY+j<numCols) {
              int index = numCols * (cornerX+i) + (cornerY+j);
							uchar4 color = d_img[index];
							sumRGB[0] += (int) color.x;
							sumRGB[1] += (int) color.y;
							sumRGB[2] += (int) color.z;
							accumulator++;
						}
					}
				}



				if(accumulator == 0){
					sumRGB[0] = pixelColor.x;
					sumRGB[1] = pixelColor.y;
					sumRGB[2] = pixelColor.z;
				}else{
					sumRGB[0] = sumRGB[0] / accumulator;
					sumRGB[1] = sumRGB[1] / accumulator;
					sumRGB[2] = sumRGB[2] / accumulator;
				}

				uchar4 color;
				color.x = sumRGB[0];
				color.y = sumRGB[1];
				color.z = sumRGB[2];


				d_blurred[pixel] = color;


			}
		}


}

int main(int argc, char** argv ){

    printf("-2");

    img = imread( argv[1], 1 );
		if ( !img.data ){
        printf("No image data \n");
        return -1;
    }

    printf("-1");
    size_t numRows =  img.rows;
    size_t numCols = img.cols;
    const size_t numPixels = numRows * numCols;

		size_t k = atoi(argv[3]);
		int THREADS = atoi(argv[4]);
    printf("0");

    uchar4 *h_img = img.ptr<uchar4>(0);
    uchar4 *h_blurred = blurred.ptr<uchar4>(0);
    uchar4 *d_img;
    uchar4 *d_blurred;
    printf("1");

    //allocate memory on the device for both input and output
    hipMalloc(&d_img, sizeof(uchar4) * numPixels);
    hipMalloc(&d_blurred, sizeof(uchar4) * numPixels);
    printf("2");

    //make sure no memory is left laying around
    hipMemset(d_blurred, 0, numPixels * sizeof(uchar4));
    printf("3");
    //copy input array to the GPU
    hipMemcpy(&d_img, &h_img, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice);
    printf("4");
		BoxBlur<<<1, THREADS>>>(d_img, d_blurred, numRows, numCols, k);
    printf("5");
    blurred = Mat(numRows, numCols, CV_8UC3, (void*)&h_blurred);

		imwrite( argv[2], blurred );

		return 0;
}
