#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include<iostream>
#include<gmpxx.h>
#include<gmp.h>
#include<vector>
#include<string>
#include<cstdio>
#include <stdio.h>

using namespace std;

__global__ void gcd_vector(int * d_out){
  int idx = threadIdx.x;
  int u = idx, v = blockDim.x;
  while ( v != 0) {
    int r = u % v;
    u = v;
    v = r;
  }
  if(u == 1){
    d_out[idx]=1;
  }
}

void set_m(mpz_t m, mpz_t p, mpz_t q)
{
  mpz_mul(m, p, q);
}

void set_phi(mpz_t phi, mpz_t p, mpz_t q)
{
  mpz_t first;
  mpz_init(first);
  mpz_sub_ui(first, p, 1);

  mpz_t second;
  mpz_init(second);
  mpz_sub_ui(second, q, 1);

  mpz_mul(phi, first, second);
}

void set_k(mpz_t k, mpz_t phi)
{
  unsigned long seed;
  mpz_t gcd;
  mpz_init(gcd);
  mpz_t n;
  mpz_init(n);
  mpz_set_str(n, "1000000", 10);
  gmp_randstate_t state;
  gmp_randinit_default (state);
  mpz_urandomm (k, state, n);
  bool abort = false;
  for(k; mpz_cmp(k, phi) <= 0; mpz_add_ui(k, k, 1)) {
      mpz_gcd(gcd, k, phi);
      if(mpz_cmp_ui(gcd, 1) == 0) return;
  }
}

void set_d(mpz_t d, mpz_t phi, mpz_t k)
{
  mpz_t i;
  mpz_init(i);


  bool abort = false;
  int integer_phi = mpz_get_si (phi);
  int integer_i = mpz_get_si (i);
  int x;
  int dummy_variable;

  for(; mpz_cmp(i, phi) <= 0; mpz_add_ui(i, i, 1)) {
    mpz_add_ui(i, i, 1);
    if(!(mpz_cmp(i, phi) <= 0)) abort = true;
    if(!abort){
      mpz_t j;
      mpz_init(j);
      mpz_set_str(j, "1", 10);

      mpz_addmul(j, i, phi);

      mpz_t res;
      mpz_init(res);

      mpz_mod(res, j, k);

      if(mpz_cmp_ui(res, 0) == 0) {

        mpz_divexact(d, j, k);

        abort = true;
        return;
      }
    }
  }
  return;
}
unsigned GCD(unsigned u, unsigned v) {
    while ( v != 0) {
        unsigned r = u % v;
        u = v;
        v = r;
    }
    return u;
}

void crack_phi(mpz_t cracked_phi, mpz_t m)
{

  mpz_t p;
  mpz_init(p);

  mpz_t gcd;
  mpz_init(gcd);

  int sum;
  int integer_m = mpz_get_si (m);

  int h_out[integer_m];
  int * d_out;

  hipMalloc((void **) &d_out, integer_m*sizeof(int));


  //launch the kernel
  gcd_vector<<<1,integer_m>>>(d_out);

  // transfer data back to host
  hipMemcpy(h_out, d_out, integer_m*sizeof(int), hipMemcpyDeviceToHost);

  for(int i=0; i<integer_m; i++){
    sum += h_out[i];
  }
  mpz_set_si(cracked_phi, sum);

}

int main(int argc, char *argv[10])
{

  string gen("generate");
  string crack("crack");




int print;

  if (!argv[1]) {
    cout << "Uso: crack <public>, decrypt <public>, encrypt <private>, generate" << endl;
    return 0;
  }

  if (argv[1] == gen){

    if (!argv[3]) {
      cout << "Se necesita especificar en el último argumento el nivel de impresión. (0/1)" << endl;
      return 0;
    }
    print =  atoi(argv[3]);

    mpz_t p;
    mpz_init(p);
    mpz_set_str(p, argv[2], 10);
    mpz_nextprime(p, p);
    if(print == 1)cout << "P: " << p << endl;

    mpz_t q;
    mpz_init(q);
    mpz_nextprime(q, p);
    if(print == 1)cout << "Q: " << q << endl << endl;

    mpz_t m;
    mpz_init(m);
    set_m(m, p, q);
    if(print == 1)cout << "m: " << m << endl;

    mpz_t phi;
    mpz_init(phi);
    set_phi(phi, p, q);
    if(print == 1)cout << "Ø(m): " << phi << endl << endl;

    mpz_t k;
    mpz_init(k);
    set_k(k, phi);
    if(print == 1)cout << "Llave pública: " << k << " " << m << endl;
    else cout  << k << " " << m << " ";
    mpz_t d;
    mpz_init(d);
    set_d(d, phi, k);
    if(print == 1)cout << "Llave privada: " << d << " " << m << endl;
    else cout << d << endl;
  } else if (argv[1] == crack) {

    if (!argv[4]) {
      cout << "Se necesita el número de hilos, abortando." << endl;
      return 0;
    }
    if (!argv[5]) {
      cout << "Se necesita especificar en el último argumento el nivel de impresión. (0/1)" << endl;
      return 0;
    }
    print =  atoi(argv[5]);

    int nProcessors = 8; //TODO
    int threads =  atoi(argv[4]);



    if(print == 1)cout << "Hilos a lanzar: " << threads << endl;


    if(print == 1)cout << endl << "Rompiendo la llave privada..." << endl;

    mpz_t k;
    mpz_init_set_str(k, argv[2], 10);

    mpz_t m;
    mpz_init_set_str(m, argv[3], 10);

    mpz_t cracked_phi;
    mpz_init(cracked_phi);

    mpz_t d;
    mpz_init(d);


    crack_phi(cracked_phi, m);

    set_d(d, cracked_phi, k);

    if(print == 1)cout << "Resultado: " << d << endl;
    else cout << d << endl;
  }

  return 0;
}
